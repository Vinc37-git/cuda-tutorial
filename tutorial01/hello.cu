
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void cuda_hello() {
    printf("Hello World from GPU!\n");
}

int main() {
    cuda_hello<<<1,1>>>();

    hipDeviceSynchronize();  // might be needed printf output.
    return 0;
}