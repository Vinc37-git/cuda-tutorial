#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>


#define N 100000000
#define MAX_ERR 1e-6

__global__ void vector_add(float *out, float *a, float *b, int n) {
    // threadIdx.x contains the index of the thread within the block
    // blockDim.x contains the size of thread block (number of threads in the thread block).

    // blockIdx.x contains the index of the block with in the grid
    // gridDim.x contains the size of the grid

    int index = (blockIdx.x * blockDim.x) + threadIdx.x;  // linearisation of index tuple
    int stride = gridDim.x * blockDim.x;  // 
    for(int i = index /*"range" for every open thread*/; i < n; i += stride /* e.g + 256*/){
        out[i] = a[i] + b[i];
    }
}

int main(){
    float *a, *b, *out;
    float *d_a, *d_b, *d_out; 

    // Allocate host memory
    a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    // Initialize host arrays
    for(int i = 0; i < N; i++){
        a[i] = 1.0f;
        b[i] = 2.0f;
    }

    // Allocate device memory
    hipMalloc((void**)&d_a, sizeof(float) * N);
    hipMalloc((void**)&d_b, sizeof(float) * N);
    hipMalloc((void**)&d_out, sizeof(float) * N);

    // Transfer data from host to device memory
    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);

    // Executing kernel 
    int threads = 1024;
    int blocks = ((N + threads) / threads);
    if (N / blocks > threads) {
        printf("Error: block dimension too small"); 
        return 0;
    }
    vector_add<<<blocks, threads>>>(d_out, d_a, d_b, N);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();  
    
    // Transfer data back to host memory
    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);

    // Verification
    for(int i = 0; i < N; i++){
        assert(fabs(out[i] - a[i] - b[i]) < MAX_ERR);
    }

    printf("PASSED\n");
    printf("First: %f and last: %f element\n", out[0], out[N-1]);

    // Deallocate device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);
    
    // Deallocate host memory
    free(a); 
    free(b); 
    free(out);
}
