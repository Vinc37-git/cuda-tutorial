#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>


#define N 100000000
#define MAX_ERR 1e-6

__global__ void vector_add(float *out, float *a, float *b, int n) {
    // threadIdx.x contains the index of the thread within the block
    // blockDim.x contains the size of thread block (number of threads in the thread block).
    int index = threadIdx.x;
    int stride = blockDim.x;
    for(int i = index /*"range" for every open thread*/; i < n; i += stride /* e.g + 256*/){
        out[i] = a[i] + b[i];
    }
}

int main(){
    float *a, *b, *out;
    float *d_a, *d_b, *d_out; 

    // Allocate host memory
    a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    // Initialize host arrays
    for(int i = 0; i < N; i++){
        a[i] = 1.0f;
        b[i] = 2.0f;
    }

    // Allocate device memory
    hipMalloc((void**)&d_a, sizeof(float) * N);
    hipMalloc((void**)&d_b, sizeof(float) * N);
    hipMalloc((void**)&d_out, sizeof(float) * N);

    // Transfer data from host to device memory
    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);

    // Executing kernel 
    vector_add<<<1/*one block*/, 256/*256 parallel threads*/>>>(d_out, d_a, d_b, N);
    
    // Transfer data back to host memory
    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);

    // Verification
    for(int i = 0; i < N; i++){
        assert(fabs(out[i] - a[i] - b[i]) < MAX_ERR);
    }

    printf("PASSED\n");

    // Deallocate device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    // Deallocate host memory
    free(a); 
    free(b); 
    free(out);
}
